#include "hip/hip_runtime.h"
// cpp header files
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

// my header files
#include "../inc/config.h"
#include "../inc/array.h"
#include "../inc/config.h"
#include "../inc/cuheun.h"
#include "../inc/cumalloc.h"
#include "../inc/cudefine.h"
#include "../inc/cufields.h"
#include "../inc/cuthermal.h"
#include "../inc/neighbourlist.h"

namespace cufields {

	__global__ void uniform(int N, double x, double y, double z, double *Hapx, double *Hapy, double *Hapz){

		const int i = blockDim.x*blockIdx.x + threadIdx.x;

		if (i < N){
			Hapx[i] = x;
			Hapy[i] = y;
			Hapz[i] = z;  
		}

	}

	__global__ void uniform_staggered(int N, double x, double y, double z, double *Hapx, double *Hapy, double *Hapz){

		const int i = blockDim.x*blockIdx.x + threadIdx.x;

		if (i < N){
			if (( i % 4 == 0) || (i % 4 == 2)) {
				Hapx[i] = x;
				Hapy[i] = y;
				Hapz[i] = z; 
			} 
			if (( i % 4 == 1) || (i % 4 == 3)) {
				Hapx[i] = -1*x;
				Hapy[i] = -1*y;
				Hapz[i] = -1*z; 
			} 
		}

	}

	__global__ void square_pulse(int N, double time, double start_time, double end_time, double height, double *Hapx, double *Hapy, double *Hapz){

		const int i = blockDim.x*blockIdx.x + threadIdx.x;

		if (i < N){

			if ((time >= start_time) && (time < end_time)){
				Hapx[i] = height;
				Hapy[i] = 0.0;
				Hapz[i] = 0.0;  
			}
			else {
				Hapx[i] = 0.0;
				Hapy[i] = 0.0;
				Hapz[i] = 0.0; 
			}

		}

	}

		__global__ void square_pulse_staggered(int N, double time, double start_time, double end_time, double height, double *Hapx, double *Hapy, double *Hapz){

		const int i = blockDim.x*blockIdx.x + threadIdx.x;

		if (i < N){

			if ((time >= start_time) && (time < end_time)){
					if (( i % 4 == 0) || (i % 4 == 2)) {
						Hapx[i] = 0.0;
						Hapy[i] = height;
						Hapz[i] = 0.0;  
					}
					else if (( i % 4 == 1) || (i % 4 == 3)) {
						Hapx[i] = 0.0;
						Hapy[i] = -1.0 * height;
						Hapz[i] = 0.0;  
					}
			}
			else {
				Hapx[i] = 0.0;
				Hapy[i] = 0.0;
				Hapz[i] = 0.0; 
			}

		}

	}

	__global__ void gaussian_pulse(int N, double time, double height, double std_dev, double centre_pos,  double *Hapx, double *Hapy, double *Hapz){

		const int i = blockDim.x*blockIdx.x + threadIdx.x;

		double gauss;
		gauss = height * exp(-1 * (((time - centre_pos) * (time - centre_pos))/(2 * std_dev * std_dev)));

		if (i < N){

			Hapx[i] = gauss;
			Hapy[i] = 0.0;
			Hapz[i] = 0.0; 

		}


	}

		__global__ void gaussian_pulse_staggered(int N, double time, double height, double std_dev, double centre_pos,  double *Hapx, double *Hapy, double *Hapz){

		const int i = blockDim.x*blockIdx.x + threadIdx.x;

		double gauss;
		gauss = height * exp(-1 * (((time - centre_pos) * (time - centre_pos))/(2 * std_dev * std_dev)));

		if (i < N){
			if (( i % 4 == 0) || (i % 4 == 2)) {
				Hapx[i] = 0.0;
				Hapy[i] = gauss;
				Hapz[i] = 0.0;  
			}
			else if (( i % 4 == 1) || (i % 4 == 3)) {
				Hapx[i] = 0.0;
				Hapy[i] = -1.0 * gauss;
				Hapz[i] = 0.0;  
			}
		}
	}

	__global__ void multi_cycle_pulse(int N, double time, double height, double std_dev, double centre_pos, double freq, double *Hapx, double *Hapy, double *Hapz){

		const int i = blockDim.x*blockIdx.x + threadIdx.x;

		double gauss;
		gauss = height * exp(-1 * (((time - centre_pos) * (time - centre_pos))/(2 * std_dev * std_dev))) * sin(2*M_PI*freq*(time - centre_pos));

		if (i < N){

			Hapx[i] = gauss;
			Hapy[i] = 0.0;
			Hapz[i] = 0.0; 

		}


	}

	__global__ void multi_cycle_pulse_staggered(int N, double time, double height, double std_dev, double centre_pos, double freq, double *Hapx, double *Hapy, double *Hapz){

		const int i = blockDim.x*blockIdx.x + threadIdx.x;

		double gauss;
		gauss = height * exp(-1 * (((time - centre_pos) * (time - centre_pos))/(2 * std_dev * std_dev))) * sin(2*M_PI*freq*(time - centre_pos));

		if (i < N){
			if (( i % 4 == 0) || (i % 4 == 2)) {
				Hapx[i] = 0.0;
				Hapy[i] = 0.0;
				Hapz[i] = gauss;  
			}
			else if (( i % 4 == 1) || (i % 4 == 3)) {
				Hapx[i] = 0.0;
				Hapy[i] = 0.0;
				Hapz[i] = -1.0 * gauss;  
			}
		}


	}

	__global__ void sine_pulse(int N, double time, double height, double freq, double *Hapx, double *Hapy, double *Hapz){

		const int i = blockDim.x*blockIdx.x + threadIdx.x;

		double gauss;
		gauss = height * sin(2*M_PI*freq*time);

		if (i < N){
			if (( i % 4 == 0) || (i % 4 == 2)) {
				Hapx[i] = 0.0;
				Hapy[i] = gauss;
				Hapz[i] = 0.0;  
			}
			else if (( i % 4 == 1) || (i % 4 == 3)) {
				Hapx[i] = 0.0;
				Hapy[i] = -1.0 * gauss;
				Hapz[i] = 0.0;  
			}
		}


	}

	void testing(int i){

		Array<double> testingx;
		Array<double> testingy, testingz;
		testingx.resize(params::Nspins);
		// testingy.resize(params::Nspins);
		// testingz.resize(params::Nspins);

		// CUDA_CALL(hipMemcpy(testingx.ptr(), cuglob::Hapz, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
		// CUDA_CALL(hipMemcpy(testingy.ptr(), Hapy, sizeof(double) * params::Lz, hipMemcpyDeviceToHost));
		// CUDA_CALL(hipMemcpy(testingz.ptr(), Hapz, sizeof(double) * params::Lz, hipMemcpyDeviceToHost));

	    std::cout << i << " ";
	    for (int a = 0; a < 5; a++){
		    std::cout << testingx(a) << " ";	
	    }
	    std::cout << std::endl;
	}

}
