// cpp header files
#include <hip/hip_runtime.h>
#include <sstream>
#include <hiprand.h>
#include <iostream>
#include <hip/hip_runtime.h>

// my header files
#include "../inc/geom.h"
#include "../inc/neighbourlist.h"
#include "../inc/fields.h"
#include "../inc/config.h"
#include "../inc/cuheun.h"
#include "../inc/cufuncs.h"
#include "../inc/cufields.h"
#include "../inc/cumalloc.h"
#include "../inc/cuthermal.h"



namespace cufuncs {

	int threadsperblock;
	int bpg;
	int nspinsdw;

	void init_device_vars(){
		threadsperblock = 256;
        bpg = (params::Nspins + threadsperblock - 1) / threadsperblock;
		nspinsdw = params::Ly*params::Lz*params::Nq;

		//testing for hedgehog
		nspinsdw = (2*(params::Lx*params::Ly) + 2*(params::Lx*(params::Ly-2)) + 2*((params::Lx-2)*(params::Ly-2)))*params::Nq;
	}

	void cuDomainWall(){
		cuheun::cuFixSpins1<<<bpg,threadsperblock>>>(nspinsdw, cuglob::dlw, cuglob::drw, cuglob::dsurfx, cuglob::dsurfy, cuglob::dsurfz, cuglob::dSx1d, cuglob::dSy1d, cuglob::dSz1d);
	}

	void cuFields(std::string type, double time, double start_time, double end_time, double height){
		if (type == "Uniform"){
			cufields::uniform<<<bpg,threadsperblock>>>(params::Nspins, fields::cuniform[0], fields::cuniform[1], fields::cuniform[2], cuglob::Hapx, cuglob::Hapy, cuglob::Hapz);
		}
		else if (type == "Uniform_Staggered"){
			cufields::uniform_staggered<<<bpg,threadsperblock>>>(params::Nspins, fields::cuniform[0], fields::cuniform[1], fields::cuniform[2], cuglob::Hapx, cuglob::Hapy, cuglob::Hapz);
		}
		else if (type == "Square_Pulse"){
			cufields::square_pulse<<<bpg,threadsperblock>>>(params::Nspins, time, start_time, end_time, height, cuglob::Hapx, cuglob::Hapy, cuglob::Hapz);
		}
		else if (type == "Gaussian_Pulse"){
			cufields::gaussian_pulse<<<bpg,threadsperblock>>>(params::Nspins, time, fields::height, fields::std_dev, fields::centre_pos, cuglob::Hapx, cuglob::Hapy, cuglob::Hapz);
		}
		else if (type == "Multi_Cycle_Pulse"){
			cufields::multi_cycle_pulse<<<bpg,threadsperblock>>>(params::Nspins, time, fields::height, fields::std_dev, fields::centre_pos, fields::freq, cuglob::Hapx, cuglob::Hapy, cuglob::Hapz);
		}
		else if (type == "Square_Pulse_Staggered"){
			cufields::square_pulse_staggered<<<bpg,threadsperblock>>>(params::Nspins, time, start_time, end_time, height, cuglob::Hapx, cuglob::Hapy, cuglob::Hapz);
		}
		else if (type == "Gaussian_Pulse_Staggered"){
			cufields::gaussian_pulse_staggered<<<bpg,threadsperblock>>>(params::Nspins, time, fields::height, fields::std_dev, fields::centre_pos, cuglob::Hapx, cuglob::Hapy, cuglob::Hapz);
		}
		else if (type == "Multi_Cycle_Pulse_Staggered"){
			cufields::multi_cycle_pulse_staggered<<<bpg,threadsperblock>>>(params::Nspins, time, fields::height, fields::std_dev, fields::centre_pos, fields::freq, cuglob::Hapx, cuglob::Hapy, cuglob::Hapz);
		}
		else if (type == "Sine_Pulse"){
			cufields::sine_pulse<<<bpg,threadsperblock>>>(params::Nspins, time, fields::height, fields::freq, cuglob::Hapx, cuglob::Hapy, cuglob::Hapz);
		}
		else {
			std::cout << "ERROR: Unkown field type: " << type << std::endl;
			exit(0);		
		}
		
	}
	

	void cuTemperature(std::string type, double time, double ttm_start){
		if (type == "ttm"){
			if (time < ttm_start){

				cuthermal::ttf<<<bpg,threadsperblock>>>(time, params::Nspins, cuthermal::dtfa, cuthermal::Te, cuthermal::dzlayer);
			}
			else {
				cuthermal::ttm<<<bpg,threadsperblock>>>(time - ttm_start, params::Lz, cuthermal::Te, cuthermal::Tp, cuthermal::P_it);
				cuthermal::ttf<<<bpg,threadsperblock>>>(time - ttm_start, params::Nspins, cuthermal::dtfa, cuthermal::Te, cuthermal::dzlayer);
			}
		}
		else if (type == "constant"){
			cuthermal::ttf<<<bpg,threadsperblock>>>(time - ttm_start, params::Nspins, cuthermal::dtfa, cuthermal::Te, cuthermal::dzlayer);
		}
		else if (type == "uniform_gradient"){
			cuthermal::ttfg<<<bpg,threadsperblock>>>(time - ttm_start, params::Nspins, cuthermal::dtfa, cuthermal::Te, cuthermal::dxlayer, params::temp_gradient);
		}
	}
	
	void cuRotation(){
		cuheun::cuRotfun<<<bpg,threadsperblock>>>(params::Nspins, cuglob::dSx1d, cuglob::dSy1d, cuglob::dSz1d); 
	}

	void integration(double time){
		cuheun::cuHeun1<<<bpg,threadsperblock>>>(cuglob::djind, neigh::nsimspin, time, cuglob::dsimspin, cuthermal::dtfa, cuthermal::gvalsx, cuthermal::gvalsy, cuthermal::gvalsz, cuglob::dx_adj, cuglob::dadjncy, cuheun::Htx, cuheun::Hty, cuheun::Htz, cuglob::dSx1d, cuglob::dSy1d, cuglob::dSz1d, cuglob::dJx_new, cuglob::dJy_new, cuglob::dJz_new, cuglob::Hapx, cuglob::Hapy, cuglob::Hapz, cuheun::DelSx,  cuheun::DelSy, cuheun::DelSz, cuheun::Sdashnx, cuheun::Sdashny, cuheun::Sdashnz);
		cuheun::cuHeun2<<<bpg,threadsperblock>>>(cuglob::djind, neigh::nsimspin, time, cuglob::dsimspin, cuglob::dx_adj, cuglob::dadjncy, cuheun::Htx, cuheun::Hty, cuheun::Htz, cuglob::dSx1d, cuglob::dSy1d, cuglob::dSz1d, cuglob::dJx_new, cuglob::dJy_new, cuglob::dJz_new, cuglob::Hapx, cuglob::Hapy, cuglob::Hapz, cuheun::DelSx, cuheun::DelSy, cuheun::DelSz, cuheun::Sdashnx, cuheun::Sdashny, cuheun::Sdashnz);
	}

}
