#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../inc/array.h"
#include "../inc/neighbourlist.h"
#include "../inc/config.h"
#include "../inc/cuheun.h"
#include "../inc/cuthermal.h"
#include "../inc/cudefine.h"
#include "../inc/fields.h"
#include "../inc/geom.h"
#include "../inc/spins.h"
#include "../inc/defines.h"
#include <cstring> 


namespace cuglob {

	double *dSx1d, *dSy1d, *dSz1d;
	double *Hapx, *Hapy, *Hapz;
	double *dJx, *dJy, *dJz;
	int *dlw, *drw;
	int *dx_adj, *dadjncy;
	Array<double> pJx, pJy, pJz;
	Array<int> px_adj, padjncy;

	//testing for hedgehog
	double *dsurfx, *dsurfy, *dsurfz;

	int device = 0;

	void device_info(){
		hipGetDevice(&device);
		struct hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device);
		TITLE("CUDA DEVICE PROPERTIES");
		std::cout.width(75); std::cout << std::left << "Device name:"; std::cout << properties.name << std::endl;
		std::cout.width(75); std::cout << std::left << "Memory Clock Rate (KHz):"; std::cout << properties.memoryClockRate << std::endl;
    	std::cout.width(75); std::cout << std::left << "Memory Bus Width (bits):"; std::cout << properties.memoryBusWidth << std::endl;
    	std::cout.width(75); std::cout << std::left << "Peak Memory Bandwidth (GB/s):"; std::cout << 2.0*properties.memoryClockRate*(properties.memoryBusWidth/8)/1.0e6 << std::endl;
		std::cout.width(75); std::cout << std::left << "multiprocessors:"; std::cout << properties.multiProcessorCount << std::endl;
		std::cout.width(75); std::cout << std::left << "max threads per processor:"; std::cout << properties.maxThreadsPerMultiProcessor << std::endl;
		std::cout.width(75); std::cout << std::left << "max threads per block:"; std::cout << properties.maxThreadsPerBlock << std::endl;	
	}

	void clear_memory(){
		CUDA_CALL(hipFree(dSx1d));
		CUDA_CALL(hipFree(dSy1d));
		CUDA_CALL(hipFree(dSz1d));
		CUDA_CALL(hipFree(cuheun::Sdashnx));
		CUDA_CALL(hipFree(cuheun::Sdashny));
		CUDA_CALL(hipFree(cuheun::Sdashnz));
		CUDA_CALL(hipFree(cuheun::DelSx));
		CUDA_CALL(hipFree(cuheun::DelSy));
		CUDA_CALL(hipFree(cuheun::DelSz));
		CUDA_CALL(hipFree(cuheun::Htx));
		CUDA_CALL(hipFree(cuheun::Hty));
		CUDA_CALL(hipFree(cuheun::Htz));
		CUDA_CALL(hipFree(Hapx));
		CUDA_CALL(hipFree(Hapy));
		CUDA_CALL(hipFree(Hapz));
		CUDA_CALL(hipFree(dJx));
		CUDA_CALL(hipFree(dJy));
		CUDA_CALL(hipFree(dJz));
		CUDA_CALL(hipFree(dx_adj));
		CUDA_CALL(hipFree(dadjncy));
		CUDA_CALL(hipFree(cuthermal::gvalsx));
		CUDA_CALL(hipFree(cuthermal::gvalsy));
		CUDA_CALL(hipFree(cuthermal::gvalsz));
		CUDA_CALL(hipFree(cuthermal::Te));
		CUDA_CALL(hipFree(cuthermal::P_it));
		CUDA_CALL(hipFree(cuthermal::Tp));
		CUDA_CALL(hipFree(dlw));
		CUDA_CALL(hipFree(drw));
		INFO_OUT("memory deallocated on GPU device: ", "success" << std::endl);
	}


	void allocate_heun_memory(){

		//Device spin variables
		CUDA_CALL(hipMalloc((void**)&dSx1d, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMalloc((void**)&dSy1d, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMalloc((void**)&dSz1d, sizeof(double)*params::Nspins));

		//Device Heun variables
		CUDA_CALL(hipMalloc((void**)&cuheun::Sdashnx, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Sdashnx, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::Sdashny, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Sdashny, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::Sdashnz, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Sdashnz, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::DelSx, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::DelSx, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::DelSy, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::DelSy, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::DelSz, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::DelSz, 0.0, sizeof(double) * params::Nspins));

		// Stochastic Magnetic Field
		CUDA_CALL(hipMalloc((void**)&cuheun::Htx, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Htx, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::Hty, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Hty, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::Htz, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Htz, 0.0, sizeof(double) * params::Nspins));

		//external field
		CUDA_CALL(hipMalloc((void**)&Hapx, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(Hapx, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&Hapy, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(Hapy, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&Hapz, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(Hapz, 0.0, sizeof(double) * params::Nspins));

		// Random number arrays
		CUDA_CALL(hipMalloc((void**)&cuthermal::gvalsx, sizeof(float)*params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuthermal::gvalsy, sizeof(float)*params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuthermal::gvalsz, sizeof(float)*params::Nspins));
		CUDA_CALL(hipMemset(cuthermal::gvalsx, 0.0, sizeof(float) * params::Nspins));
		CUDA_CALL(hipMemset(cuthermal::gvalsy, 0.0, sizeof(float) * params::Nspins));
		CUDA_CALL(hipMemset(cuthermal::gvalsz, 0.0, sizeof(float) * params::Nspins));

		// Jij matrices
		CUDA_CALL(hipMalloc((void**)&dJx, sizeof(double)*neigh::Jijx_prime.size()));
		CUDA_CALL(hipMalloc((void**)&dJy, sizeof(double)*neigh::Jijx_prime.size()));
		CUDA_CALL(hipMalloc((void**)&dJz, sizeof(double)*neigh::Jijx_prime.size()));
		CUDA_CALL(hipMalloc((void**)&dx_adj, sizeof(int)*neigh::x_adj.size()));
		CUDA_CALL(hipMalloc((void**)&dadjncy, sizeof(int)*neigh::adjncy.size()));

		// Domain Wall arrays
		CUDA_CALL(hipMalloc((void**)&dlw, sizeof(int)*geom::lw.size()));
		CUDA_CALL(hipMalloc((void**)&drw, sizeof(int)*geom::rw.size()));
		CUDA_CALL(hipMemset(dlw, 0.0, sizeof(int) * geom::lw.size()));
		CUDA_CALL(hipMemset(drw, 0.0, sizeof(int) * geom::rw.size()));

		//testing for hedgehog
		CUDA_CALL(hipMalloc((void**)&dsurfx, sizeof(double)*geom::surfx.size()));
		CUDA_CALL(hipMalloc((void**)&dsurfy, sizeof(double)*geom::surfy.size()));
		CUDA_CALL(hipMalloc((void**)&dsurfz, sizeof(double)*geom::surfz.size()));
		CUDA_CALL(hipMemset(dsurfx, 0.0, sizeof(double) * geom::surfx.size()));
		CUDA_CALL(hipMemset(dsurfy, 0.0, sizeof(double) * geom::surfy.size()));
		CUDA_CALL(hipMemset(dsurfz, 0.0, sizeof(double) * geom::surfz.size()));

		// Temperature arrays
		CUDA_CALL(hipMalloc((void**)&cuthermal::Te, sizeof(double)*params::Lz));
		CUDA_CALL(hipMalloc((void**)&cuthermal::Tp, sizeof(double)*params::Lz));
		CUDA_CALL(hipMalloc((void**)&cuthermal::P_it, sizeof(double)*params::Lz));
		CUDA_CALL(hipMalloc((void**)&cuthermal::dzlayer, sizeof(int)*geom::zlayer.size()));

		CUDA_CALL(hipMemset(cuthermal::Te, 0.0, sizeof(double)*params::Lz));
		CUDA_CALL(hipMemset(cuthermal::Tp, 0.0, sizeof(double)*params::Lz));
		CUDA_CALL(hipMemset(cuthermal::P_it, 0.0, sizeof(double)*params::Lz));
		CUDA_CALL(hipMalloc((void**)&cuthermal::dtfa, sizeof(double)* params::Nspins));
		CUDA_CALL(hipMemset(cuthermal::dtfa, 0.0, sizeof(double) * params::Nspins)); 


		// clear memory at exit of program
		atexit(clear_memory);

	}

	void copy_temp_to_device(double equilibium_temp){
		Array<double> rTe; 
		Array<double> rTp;

		rTe.resize(params::Lz);
		rTp.resize(params::Lz);
		for (int i = 0; i < params::Lz; i++){
			rTe[i] = equilibium_temp;
			rTp[i] = equilibium_temp;
		}

		CUDA_CALL(hipMemcpy(cuthermal::Te, rTe.ptr(), sizeof(double) * params::Lz, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(cuthermal::Tp, rTp.ptr(), sizeof(double) * params::Lz, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(geom::zlayer.ptr(), cuthermal::dzlayer, sizeof(int) * params::Nspins, hipMemcpyDeviceToHost));

	}

	void copy_field_to_device(){
		CUDA_CALL(hipMemcpy(Hapx, fields::H_appx.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(Hapy, fields::H_appy.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(Hapz, fields::H_appz.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
	}	

	void copy_jij_to_device(){

		//resize 1d arrays
		pJx.resize(neigh::Jijx_prime.size());
		pJy.resize(neigh::Jijy_prime.size());
		pJz.resize(neigh::Jijz_prime.size());	

		px_adj.resize(neigh::x_adj.size());
		padjncy.resize(neigh::adjncy.size());
		
		for (int a = 0; a < neigh::Jijx_prime.size(); a++){
			pJx(a) = neigh::Jijx_prime[a];
			pJy(a) = neigh::Jijy_prime[a];
			pJz(a) = neigh::Jijz_prime[a];
		}

		for (int a = 0; a < neigh::x_adj.size(); a++){
			px_adj(a) = neigh::x_adj[a];
		}

		for (int a = 0; a < neigh::adjncy.size(); a++){
			padjncy(a) = neigh::adjncy[a];
		}		

		CUDA_CALL(hipMemcpy(dJx, pJx.ptr(), sizeof(double) * neigh::Jijx_prime.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dJy, pJy.ptr(), sizeof(double) * neigh::Jijy_prime.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dJz, pJz.ptr(), sizeof(double) * neigh::Jijz_prime.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dx_adj, px_adj.ptr(), sizeof(int) * (neigh::x_adj.size()), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dadjncy, padjncy.ptr(), sizeof(int) * (neigh::adjncy.size()), hipMemcpyHostToDevice));

	}

	void copy_spins_to_device(){
		CUDA_CALL(hipMemcpy(dSx1d, spins::sx1d.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dSy1d, spins::sy1d.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dSz1d, spins::sz1d.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
	}

	void copy_dw_to_device(){
		CUDA_CALL(hipMemcpy(dlw, geom::lw.ptr(), sizeof(int) * geom::lw.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(drw, geom::rw.ptr(), sizeof(int) * geom::rw.size(), hipMemcpyHostToDevice));

		//testing for hedgehog
		CUDA_CALL(hipMemcpy(dsurfx, geom::surfx.ptr(), sizeof(double) * geom::surfx.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dsurfy, geom::surfy.ptr(), sizeof(double) * geom::surfy.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dsurfz, geom::surfz.ptr(), sizeof(double) * geom::surfz.size(), hipMemcpyHostToDevice));
	}

	void copy_spins_to_host(){
		CUDA_CALL(hipMemcpy(spins::sx1d.ptr(), dSx1d, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(spins::sy1d.ptr(), dSy1d, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(spins::sz1d.ptr(), dSz1d, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
	}



}
