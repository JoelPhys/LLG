#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "../inc/array.h"
#include "../inc/NeighbourList.h"
#include "../inc/params1.h"
#include "../inc/cuheun.h"
#include "../inc/cuthermal.h"
#include "../inc/cudefine.h"

namespace cuglob {

	double *dSx1d, *dSy1d, *dSz1d;
	double *Hapx, *Hapy, *Hapz;
	double *dJx, *dJy, *dJz;
	int *dx_adj, *dadjncy;
	double *dtfa;
	Array<double> pJx, pJy, pJz;
	Array<int> px_adj, padjncy;

	int device = 0;

	void device_info(){
		hipGetDevice(&device);
		struct hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device);
		std::cout << "using " << properties.multiProcessorCount << " multiprocessors" << std::endl;
		std::cout << "max threads per processor " << properties.maxThreadsPerMultiProcessor << std::endl;
		std::cout << "max threads per block " << properties.maxThreadsPerBlock << std::endl;	
	}

	void clear_memory(){
		CUDA_CALL(hipFree(dSx1d));
		CUDA_CALL(hipFree(dSy1d));
		CUDA_CALL(hipFree(dSz1d));
		CUDA_CALL(hipFree(cuheun::Sdashnx));
		CUDA_CALL(hipFree(cuheun::Sdashny));
		CUDA_CALL(hipFree(cuheun::Sdashnz));
		CUDA_CALL(hipFree(cuheun::DelSx));
		CUDA_CALL(hipFree(cuheun::DelSy));
		CUDA_CALL(hipFree(cuheun::DelSz));
		CUDA_CALL(hipFree(cuheun::Htx));
		CUDA_CALL(hipFree(cuheun::Hty));
		CUDA_CALL(hipFree(cuheun::Htz));
		CUDA_CALL(hipFree(Hapx));
		CUDA_CALL(hipFree(Hapy));
		CUDA_CALL(hipFree(Hapz));
		CUDA_CALL(hipFree(dJx));
		CUDA_CALL(hipFree(dJy));
		CUDA_CALL(hipFree(dJz));
		CUDA_CALL(hipFree(dx_adj));
		CUDA_CALL(hipFree(dadjncy));
		CUDA_CALL(hipFree(cuthermal::gvalsx));
		CUDA_CALL(hipFree(cuthermal::gvalsy));
		CUDA_CALL(hipFree(cuthermal::gvalsz));
		std::cout << "memory deallocated on device" << std::endl;
	}


	void allocate_heun_memory(){

		//Device spin variables
		CUDA_CALL(hipMalloc((void**)&dSx1d, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMalloc((void**)&dSy1d, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMalloc((void**)&dSz1d, sizeof(double)*params::Nspins));

		//Device Heun variables
		CUDA_CALL(hipMalloc((void**)&cuheun::Sdashnx, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Sdashnx, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::Sdashny, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Sdashny, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::Sdashnz, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Sdashnz, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::DelSx, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::DelSx, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::DelSy, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::DelSy, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::DelSz, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::DelSz, 0.0, sizeof(double) * params::Nspins));


		// Stochastic Magnetic Field
                CUDA_CALL(hipMalloc((void**)&cuheun::Htx, sizeof(double)*params::Nspins));
                CUDA_CALL(hipMemset(cuheun::Htx, 0.0, sizeof(double) * params::Nspins));
                CUDA_CALL(hipMalloc((void**)&cuheun::Hty, sizeof(double)*params::Nspins));
                CUDA_CALL(hipMemset(cuheun::Hty, 0.0, sizeof(double) * params::Nspins));
                CUDA_CALL(hipMalloc((void**)&cuheun::Htz, sizeof(double)*params::Nspins));
                CUDA_CALL(hipMemset(cuheun::Htz, 0.0, sizeof(double) * params::Nspins));

		//external field
		CUDA_CALL(hipMalloc((void**)&Hapx, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(Hapx, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&Hapy, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(Hapy, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&Hapz, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(Hapz, 0.0, sizeof(double) * params::Nspins));

		// Random number arrays
		CUDA_CALL(hipMalloc((void**)&cuthermal::gvalsx, sizeof(float)*params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuthermal::gvalsy, sizeof(float)*params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuthermal::gvalsz, sizeof(float)*params::Nspins));
		CUDA_CALL(hipMemset(cuthermal::gvalsx, 0.0, sizeof(float) * params::Nspins));
		CUDA_CALL(hipMemset(cuthermal::gvalsy, 0.0, sizeof(float) * params::Nspins));
		CUDA_CALL(hipMemset(cuthermal::gvalsz, 0.0, sizeof(float) * params::Nspins));


		//thermal array
		CUDA_CALL(hipMalloc((void**)&dtfa, sizeof(double)* params::Nspins));
		CUDA_CALL(hipMemset(dtfa, 0.0, sizeof(double) * params::Nspins));


		// Jij matrices
		CUDA_CALL(hipMalloc((void**)&dJx, sizeof(double)*neigh::Jijx_prime.size()));
		CUDA_CALL(hipMalloc((void**)&dJy, sizeof(double)*neigh::Jijx_prime.size()));
		CUDA_CALL(hipMalloc((void**)&dJz, sizeof(double)*neigh::Jijx_prime.size()));
		CUDA_CALL(hipMalloc((void**)&dx_adj, sizeof(int)*neigh::x_adj.size()));
		CUDA_CALL(hipMalloc((void**)&dadjncy, sizeof(int)*neigh::adjncy.size()));

		// clear memory at exit of program
		atexit(clear_memory);

	}

	void copy_field_to_device(){
		CUDA_CALL(hipMemcpy(Hapx, params::H_appx.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(Hapy, params::H_appy.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(Hapz, params::H_appz.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
	}

	void copy_thermal_to_device(double Thermal_Fluct){
		Array<double> tfa;
		tfa.resize(params::Nspins);
		for (int a = 0; a < params::Nspins; a++){
			tfa(a) = Thermal_Fluct;
		}

		CUDA_CALL(hipMemcpy(dtfa, tfa.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));    
	}	

	void copy_jij_to_device(){

		//resize 1d arrays
		pJx.resize(neigh::Jijx_prime.size());
		pJy.resize(neigh::Jijy_prime.size());
		pJz.resize(neigh::Jijz_prime.size());	

		px_adj.resize(neigh::x_adj.size());
		padjncy.resize(neigh::adjncy.size());
		
		for (int a = 0; a < neigh::Jijx_prime.size(); a++){
			pJx(a) = neigh::Jijx_prime[a];
			pJy(a) = neigh::Jijy_prime[a];
			pJz(a) = neigh::Jijz_prime[a];
		}

		for (int a = 0; a < neigh::x_adj.size(); a++){
			px_adj(a) = neigh::x_adj[a];
		}

		for (int a = 0; a < neigh::adjncy.size(); a++){
			padjncy(a) = neigh::adjncy[a];
		}		

		CUDA_CALL(hipMemcpy(dJx, pJx.ptr(), sizeof(double) * neigh::Jijx_prime.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dJy, pJy.ptr(), sizeof(double) * neigh::Jijy_prime.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dJz, pJz.ptr(), sizeof(double) * neigh::Jijz_prime.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dx_adj, px_adj.ptr(), sizeof(int) * (neigh::x_adj.size()), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dadjncy, padjncy.ptr(), sizeof(int) * (neigh::adjncy.size()), hipMemcpyHostToDevice));

	}

	void copy_spins_to_device(){
		CUDA_CALL(hipMemcpy(dSx1d, neigh::Sx1d.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dSy1d, neigh::Sy1d.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dSz1d, neigh::Sz1d.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
	}

	void copy_spins_to_host(){
		CUDA_CALL(hipMemcpy(neigh::Sx1d.ptr(), dSx1d, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(neigh::Sy1d.ptr(), dSy1d, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(neigh::Sz1d.ptr(), dSz1d, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
	}



}
