#include "hip/hip_runtime.h"
// cpp header files
#include <hip/hip_runtime.h>
#include <cstring>
#include <hiprand.h>
#include <iostream>
#include <hip/hip_runtime.h>

// my header files
#include "../inc/geom.h"
#include "../inc/array.h"
#include "../inc/spins.h"
#include "../inc/fields.h"
#include "../inc/config.h"
#include "../inc/cuheun.h"
#include "../inc/defines.h"
#include "../inc/cudefine.h"
#include "../inc/cuthermal.h"
#include "../inc/neighbourlist.h"


namespace cuglob {

	double *dSx1d, *dSy1d, *dSz1d;
	double *Hapx, *Hapy, *Hapz;
	double *dEx, *dEy, *dEz;
	double *dJx, *dJy, *dJz;
	int *dlw, *drw;
	int *dx_adj, *dadjncy;
	int *dsimspin;
	int *dsublat_sites;

	//testing for hedgehog
	double *dsurfx, *dsurfy, *dsurfz;

	// Damping
	double *c_lambda, *c_lambdap;
		
	// Unaxiail anisotropy array
	double *danix, *daniy, *daniz;

	//testing
	double *dJx_new, *dJy_new, *dJz_new;
	int *djind;

	//gpu variables
	int tpb;
	int bpg;
	int device = 0;

	void device_info(){
		hipGetDevice(&device);
		struct hipDeviceProp_t properties;
		hipGetDeviceProperties(&properties, device);
		TITLE("CUDA DEVICE PROPERTIES");
		INFO_OUT("Device name:", properties.name);
		INFO_OUT("Memory Clock Rate (KHz):", properties.memoryClockRate);
    	INFO_OUT("Memory Bus Width (bits):", properties.memoryBusWidth);
    	INFO_OUT("Peak Memory Bandwidth (GB/s):", 2.0*properties.memoryClockRate*(properties.memoryBusWidth/8)/1.0e6);
		INFO_OUT("multiprocessors:", properties.multiProcessorCount);
		INFO_OUT("max threads per processor:", properties.maxThreadsPerMultiProcessor);
		INFO_OUT("max threads per block:", properties.maxThreadsPerBlock);	
		tpb = properties.maxThreadsPerBlock;
		bpg = (params::Nspins + tpb - 1) / tpb;

		INFO_OUT("Simulation Threads Per Block:", tpb);
		INFO_OUT("Simulation Blocks Per grid:", bpg);
		
	}

	void clear_memory(){
		CUDA_CALL(hipFree(dSx1d));
		CUDA_CALL(hipFree(dSy1d));
		CUDA_CALL(hipFree(dSz1d));
		CUDA_CALL(hipFree(cuheun::Sdashnx));
		CUDA_CALL(hipFree(cuheun::Sdashny));
		CUDA_CALL(hipFree(cuheun::Sdashnz));
		CUDA_CALL(hipFree(cuheun::DelSx));
		CUDA_CALL(hipFree(cuheun::DelSy));
		CUDA_CALL(hipFree(cuheun::DelSz));
		CUDA_CALL(hipFree(cuheun::Htx));
		CUDA_CALL(hipFree(cuheun::Hty));
		CUDA_CALL(hipFree(cuheun::Htz));
		CUDA_CALL(hipFree(Hapx));
		CUDA_CALL(hipFree(Hapy));
		CUDA_CALL(hipFree(Hapz));
		CUDA_CALL(hipFree(dJx));
		CUDA_CALL(hipFree(dJy));
		CUDA_CALL(hipFree(dJz));
		CUDA_CALL(hipFree(dEx));
		CUDA_CALL(hipFree(dEy));
		CUDA_CALL(hipFree(dEz));
		CUDA_CALL(hipFree(dx_adj));
		CUDA_CALL(hipFree(dadjncy));
		CUDA_CALL(hipFree(dsublat_sites));
		CUDA_CALL(hipFree(cuthermal::gvalsx));
		CUDA_CALL(hipFree(cuthermal::gvalsy));
		CUDA_CALL(hipFree(cuthermal::gvalsz));
		CUDA_CALL(hipFree(cuthermal::Te));
		CUDA_CALL(hipFree(cuthermal::P_it));
		CUDA_CALL(hipFree(cuthermal::Tp));
		CUDA_CALL(hipFree(dlw));
		CUDA_CALL(hipFree(drw));
		INFO_OUT("memory deallocated on GPU device: ", "success");
	}


	void allocate_heun_memory(){

		//Device spin variables
		CUDA_CALL(hipMalloc((void**)&dSx1d, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMalloc((void**)&dSy1d, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMalloc((void**)&dSz1d, sizeof(double)*params::Nspins));

		//Device Heun variables
		CUDA_CALL(hipMalloc((void**)&cuheun::Sdashnx, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Sdashnx, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::Sdashny, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Sdashny, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::Sdashnz, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Sdashnz, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::DelSx, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::DelSx, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::DelSy, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::DelSy, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::DelSz, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::DelSz, 0.0, sizeof(double) * params::Nspins));

		// Stochastic Magnetic Field
		CUDA_CALL(hipMalloc((void**)&cuheun::Htx, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Htx, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::Hty, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Hty, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuheun::Htz, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(cuheun::Htz, 0.0, sizeof(double) * params::Nspins));

		//external field
		CUDA_CALL(hipMalloc((void**)&Hapx, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(Hapx, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&Hapy, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(Hapy, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&Hapz, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(Hapz, 0.0, sizeof(double) * params::Nspins));

		//Total Energy
		CUDA_CALL(hipMalloc((void**)&dEx, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(dEx, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&dEy, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(dEy, 0.0, sizeof(double) * params::Nspins));
		CUDA_CALL(hipMalloc((void**)&dEz, sizeof(double)*params::Nspins));
		CUDA_CALL(hipMemset(dEz, 0.0, sizeof(double) * params::Nspins));

		// Random number arrays
		CUDA_CALL(hipMalloc((void**)&cuthermal::gvalsx, sizeof(float)*params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuthermal::gvalsy, sizeof(float)*params::Nspins));
		CUDA_CALL(hipMalloc((void**)&cuthermal::gvalsz, sizeof(float)*params::Nspins));
		CUDA_CALL(hipMemset(cuthermal::gvalsx, 0.0, sizeof(float) * params::Nspins));
		CUDA_CALL(hipMemset(cuthermal::gvalsy, 0.0, sizeof(float) * params::Nspins));
		CUDA_CALL(hipMemset(cuthermal::gvalsz, 0.0, sizeof(float) * params::Nspins));

		//Damping
		CUDA_CALL(hipMalloc((void**)&c_lambda,  sizeof(double)*params::Nq));
		CUDA_CALL(hipMalloc((void**)&c_lambdap, sizeof(double)*params::Nq));
		CUDA_CALL(hipMemset(c_lambda,  0.0, sizeof(double) * params::Nq));
		CUDA_CALL(hipMemset(c_lambdap, 0.0, sizeof(double) * params::Nq));

		// Unaxiail Anisotropy Constants
		CUDA_CALL(hipMalloc((void**)&danix, sizeof(double)*params::Nq));
		CUDA_CALL(hipMalloc((void**)&daniy, sizeof(double)*params::Nq));
		CUDA_CALL(hipMalloc((void**)&daniz, sizeof(double)*params::Nq));

		//Sublattice ordering
		CUDA_CALL(hipMalloc((void**)&dsublat_sites, sizeof(int)*params::Nq));
		CUDA_CALL(hipMemset(dsublat_sites,  0.0, sizeof(int) * params::Nq));

		// Jij matrices
		CUDA_CALL(hipMalloc((void**)&dJx, sizeof(double)*neigh::Jijx_prime.size()));
		CUDA_CALL(hipMalloc((void**)&dJy, sizeof(double)*neigh::Jijx_prime.size()));
		CUDA_CALL(hipMalloc((void**)&dJz, sizeof(double)*neigh::Jijx_prime.size()));
		CUDA_CALL(hipMalloc((void**)&dx_adj, sizeof(int)*neigh::x_adj.size()));
		CUDA_CALL(hipMalloc((void**)&dadjncy, sizeof(int)*neigh::adjncy.size()));

		// Domain Wall arrays
		CUDA_CALL(hipMalloc((void**)&dlw, sizeof(int)*geom::lw.size()));
		CUDA_CALL(hipMalloc((void**)&drw, sizeof(int)*geom::rw.size()));
		CUDA_CALL(hipMemset(dlw, 0.0, sizeof(int) * geom::lw.size()));
		CUDA_CALL(hipMemset(drw, 0.0, sizeof(int) * geom::rw.size()));

		//testing for hedgehog
		CUDA_CALL(hipMalloc((void**)&dsurfx, sizeof(double)*geom::surfx.size()));
		CUDA_CALL(hipMalloc((void**)&dsurfy, sizeof(double)*geom::surfy.size()));
		CUDA_CALL(hipMalloc((void**)&dsurfz, sizeof(double)*geom::surfz.size()));
		CUDA_CALL(hipMemset(dsurfx, 0.0, sizeof(double) * geom::surfx.size()));
		CUDA_CALL(hipMemset(dsurfy, 0.0, sizeof(double) * geom::surfy.size()));
		CUDA_CALL(hipMemset(dsurfz, 0.0, sizeof(double) * geom::surfz.size()));


		//testing
		CUDA_CALL(hipMalloc((void**)&dJx_new, sizeof(double)*neigh::Jijx.size()));
		CUDA_CALL(hipMalloc((void**)&dJy_new, sizeof(double)*neigh::Jijx.size()));
		CUDA_CALL(hipMalloc((void**)&dJz_new, sizeof(double)*neigh::Jijx.size()));
		CUDA_CALL(hipMalloc((void**)&djind, sizeof(int)*neigh::jind.size()));

		CUDA_CALL(hipMalloc((void**)&dsimspin, sizeof(int) * neigh::nsimspin));
		CUDA_CALL(hipMemset(dsimspin, 0.0, sizeof(int) * neigh::nsimspin));

		// Temperature arrays
		CUDA_CALL(hipMalloc((void**)&cuthermal::Te, sizeof(double)*params::Lz));
		CUDA_CALL(hipMalloc((void**)&cuthermal::Tp, sizeof(double)*params::Lz));
		CUDA_CALL(hipMalloc((void**)&cuthermal::P_it, sizeof(double)*params::Lz));
		CUDA_CALL(hipMalloc((void**)&cuthermal::dxlayer, sizeof(int)*geom::xlayer.size()));
		CUDA_CALL(hipMalloc((void**)&cuthermal::dylayer, sizeof(int)*geom::ylayer.size()));
		CUDA_CALL(hipMalloc((void**)&cuthermal::dzlayer, sizeof(int)*geom::zlayer.size()));

		CUDA_CALL(hipMalloc((void**)&cuthermal::dconst, sizeof(double)* params::Nq));
		CUDA_CALL(hipMemset(cuthermal::dconst, 0.0, sizeof(double) * params::Nq)); 

		CUDA_CALL(hipMemset(cuthermal::Te, 0.0, sizeof(double)*params::Lz));
		CUDA_CALL(hipMemset(cuthermal::Tp, 0.0, sizeof(double)*params::Lz));
		CUDA_CALL(hipMemset(cuthermal::P_it, 0.0, sizeof(double)*params::Lz));
		CUDA_CALL(hipMalloc((void**)&cuthermal::dtfa, sizeof(double)* params::Nspins));
		CUDA_CALL(hipMemset(cuthermal::dtfa, 0.0, sizeof(double) * params::Nspins)); 



		// clear memory at exit of program
		atexit(clear_memory);

	}

	void copy_damp_to_device(){
		CUDA_CALL(hipMemcpy(c_lambda,  &params::lambda[0], sizeof(double) * params::Nq, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(c_lambdap, &params::lambdaPrime[0], sizeof(double) * params::Nq, hipMemcpyHostToDevice));
	
		CUDA_CALL(hipMemcpy(danix,  &params::dxup[0], sizeof(double) * params::Nq, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(daniy,  &params::dyup[0], sizeof(double) * params::Nq, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(daniz,  &params::dzup[0], sizeof(double) * params::Nq, hipMemcpyHostToDevice));
	}


	void copy_temp_to_device(double equilibium_temp){
		Array<double> rTe; 
		Array<double> rTp;

		rTe.resize(params::Lz);
		rTp.resize(params::Lz);
		for (int i = 0; i < params::Lz; i++){
			rTe[i] = equilibium_temp;
			rTp[i] = equilibium_temp;
		}

		CUDA_CALL(hipMemcpy(cuthermal::Te, rTe.ptr(), sizeof(double) * params::Lz, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(cuthermal::Tp, rTp.ptr(), sizeof(double) * params::Lz, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(cuthermal::dxlayer, geom::xlayer.ptr(), sizeof(int) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(cuthermal::dylayer, geom::ylayer.ptr(), sizeof(int) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(cuthermal::dzlayer, geom::zlayer.ptr(), sizeof(int) * params::Nspins, hipMemcpyHostToDevice));

		// Thermal constant
		CUDA_CALL(hipMemcpy(cuthermal::dconst, &params::thermal_const[0], sizeof(double) * params::Nq, hipMemcpyHostToDevice));

	}

	void copy_energy_to_device(){
		CUDA_CALL(hipMemcpy(dEx, spins::Ex.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dEy, spins::Ey.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dEz, spins::Ez.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));

	}

	void copy_field_to_device(){
		CUDA_CALL(hipMemcpy(Hapx, fields::H_appx.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(Hapy, fields::H_appy.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(Hapz, fields::H_appz.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
	}	

	void copy_jij_to_device(){

		CUDA_CALL(hipMemcpy(dx_adj, &neigh::x_adj[0], sizeof(int) * (neigh::x_adj.size()), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dadjncy, &neigh::adjncy[0], sizeof(int) * (neigh::adjncy.size()), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dJx, &neigh::Jijx_prime[0], sizeof(double) * neigh::Jijx_prime.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dJy, &neigh::Jijy_prime[0], sizeof(double) * neigh::Jijy_prime.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dJz, &neigh::Jijz_prime[0], sizeof(double) * neigh::Jijz_prime.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dsimspin, &neigh::simspin[0], sizeof(int) * neigh::nsimspin, hipMemcpyHostToDevice));	

		//testing
		CUDA_CALL(hipMemcpy(djind, &neigh::jind[0], sizeof(int) * neigh::jind.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dJx_new, &neigh::Jijx[0], sizeof(double) * neigh::Jijx.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dJy_new, &neigh::Jijy[0], sizeof(double) * neigh::Jijy.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dJz_new, &neigh::Jijz[0], sizeof(double) * neigh::Jijz.size(), hipMemcpyHostToDevice));

	}

	void copy_spins_to_device(){
		CUDA_CALL(hipMemcpy(dSx1d, spins::sx1d.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dSy1d, spins::sy1d.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dSz1d, spins::sz1d.ptr(), sizeof(double) * params::Nspins, hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dsublat_sites, &params::sublat_sites[0], sizeof(int) * params::Nq, hipMemcpyHostToDevice));
	}

	void copy_dw_to_device(){
		CUDA_CALL(hipMemcpy(dlw, geom::lw.ptr(), sizeof(int) * geom::lw.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(drw, geom::rw.ptr(), sizeof(int) * geom::rw.size(), hipMemcpyHostToDevice));

		//testing for hedgehog
		CUDA_CALL(hipMemcpy(dsurfx, geom::surfx.ptr(), sizeof(double) * geom::surfx.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dsurfy, geom::surfy.ptr(), sizeof(double) * geom::surfy.size(), hipMemcpyHostToDevice));
		CUDA_CALL(hipMemcpy(dsurfz, geom::surfz.ptr(), sizeof(double) * geom::surfz.size(), hipMemcpyHostToDevice));
	}

	void copy_spins_to_host(){
		CUDA_CALL(hipMemcpy(spins::sx1d.ptr(), dSx1d, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(spins::sy1d.ptr(), dSy1d, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(spins::sz1d.ptr(), dSz1d, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
	}

	void copy_field_to_host(){
		CUDA_CALL(hipMemcpy(fields::H_appx.ptr(), Hapx, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(fields::H_appy.ptr(), Hapy, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(fields::H_appz.ptr(), Hapz, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
	}

	void copy_energy_to_host(){
		CUDA_CALL(hipMemcpy(spins::Ex.ptr(), dEx, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(spins::Ey.ptr(), dEy, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(spins::Ez.ptr(), dEz, sizeof(double) * params::Nspins, hipMemcpyDeviceToHost));
	}

}
