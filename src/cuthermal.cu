#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "../inc/cudefine.h"
#include "../inc/params1.h"
#include <iostream>
#include <ctime>

namespace cuthermal {

    float *gvalsx, *gvalsy, *gvalsz;
    hiprandGenerator_t gen;

    void curand_generator(){
	std::time_t result = std::time(nullptr);
	int seed = static_cast<int>(result);
    std::cout << "time since epoch  = " << result << " (s)" << std::endl;
	CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, seed));
	std::cout << "Curand Seed = " << seed << std::endl;
    }

    void gen_thermal_noise(){
        CURAND_CALL(hiprandGenerateNormal(gen, gvalsx, params::Nspins, 0.0, 1.0));
        CURAND_CALL(hiprandGenerateNormal(gen, gvalsy, params::Nspins, 0.0, 1.0));
        CURAND_CALL(hiprandGenerateNormal(gen, gvalsz, params::Nspins, 0.0, 1.0));
    }

    void destroy_generator(){
        hiprandDestroyGenerator(gen);
	std::cout << "Curand Generator Destroyed" << std::endl;
    }


}
