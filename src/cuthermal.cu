#include "hip/hip_runtime.h"
// cpp header files
#include <ctime>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

// my header files
#include "../inc/array.h"
#include "../inc/config.h"
#include "../inc/defines.h"
#include "../inc/cudefine.h"

namespace cuthermal {



    //two temperature model variables
    double gamma_e;
    double Cp;
    double kappa_0;
    double delta;
    double Gep;
    double P_0;
    double t0;
    double tau;
    int    Nz;
    double dz;
    double dt;
    double Tinit;
    double oneOvrdzdz;
    double oneOvr2dz;

    __constant__ double c_gamma_e;
	__constant__ double c_Cp;
	__constant__ double c_kappa_0;
	__constant__ double c_delta;
	__constant__ double c_Gep;
	__constant__ double c_P_0;
	__constant__ double c_t0;
	__constant__ double c_tau;
	__constant__ double c_Nz;
	__constant__ double c_dz;
    __constant__ double c_dt;
    __constant__ double c_Tinit;
    __constant__ double c_oneOvrdzdz;
    __constant__ double c_oneOvr2dz;
    __constant__ double c_thermal_const;

    double *Te, *Tp, *P_it;
	double *dtfa;
    int *dzlayer;

    // stochastic noise variables
    float *gvalsx, *gvalsy, *gvalsz;
    hiprandGenerator_t gen;


    void init_cuthermal(double equilibrium_temp){

        //two temperature model variables
        gamma_e=125.0;                //gamma_e defines the electron specific heat through, C_e = gamma_e * T_e. [J/m^3/K^2]
        Cp=3e6;                       //Specific heat of phonons. [J/m^3/K]
        kappa_0=11.0;                 //kappa_0 defines the thermal heat conductivity (kappa) through, kappa = kappa_0 * T_e/T_p [J/m/K/s]
        delta=20.0e-9;                //Penetration depth of laser. [m]
        Gep=10e17;                    //Electron-phonon coupling [ J/m^3/s/K]
        P_0=2.0e21;                   //Pump fluence prefactor, P_0. P(z,t)=P_0*exp(-((t-t0)/tau)**2)*exp(-z/delta) [ J/m^3/s]
        t0=100e-15;                   //Pump temporal offset [s]
        tau=200e-15;                   //Pump temporal full width half max [s]
        Nz=100;                       //number of unit cells in z-direction (assumed uniform heating perpendicular [unit cells in z]
        dz=0.3e-9;                    //lattice constant (or difference between planes) [m]
        dt=1e-16;                     //Timestep [s]
        Tinit=equilibrium_temp;
        oneOvrdzdz=1./(dz*dz);
        oneOvr2dz=1./(2.0*dz);
        CUDA_CALL(hipMemcpyToSymbol(*(&c_gamma_e), &gamma_e, sizeof(double)));                        
        CUDA_CALL(hipMemcpyToSymbol(*(&c_Cp), &Cp, sizeof(double)));                                  
        CUDA_CALL(hipMemcpyToSymbol(*(&c_kappa_0), &kappa_0, sizeof(double)));                        
        CUDA_CALL(hipMemcpyToSymbol(*(&c_delta), &delta, sizeof(double)));                            
        CUDA_CALL(hipMemcpyToSymbol(*(&c_Gep), &Gep, sizeof(double)));                                
        CUDA_CALL(hipMemcpyToSymbol(*(&c_P_0), &P_0, sizeof(double)));                                
        CUDA_CALL(hipMemcpyToSymbol(*(&c_t0), &t0, sizeof(double)));                                  
        CUDA_CALL(hipMemcpyToSymbol(*(&c_tau), &tau, sizeof(double)));                                
        CUDA_CALL(hipMemcpyToSymbol(*(&c_Nz), &Nz, sizeof(int)));                                     
        CUDA_CALL(hipMemcpyToSymbol(*(&c_dz), &dz, sizeof(double)));                                  
        CUDA_CALL(hipMemcpyToSymbol(*(&c_dt), &params::dt, sizeof(double)));                          
        CUDA_CALL(hipMemcpyToSymbol(*(&c_thermal_const), &params::thermal_const, sizeof(double)));
        CUDA_CALL(hipMemcpyToSymbol(*(&c_Tinit), &Tinit, sizeof(double)));
        CUDA_CALL(hipMemcpyToSymbol(*(&c_oneOvrdzdz), &oneOvrdzdz, sizeof(double)));
        CUDA_CALL(hipMemcpyToSymbol(*(&c_oneOvr2dz), &oneOvr2dz, sizeof(double)));
    }

    void destroy_generator(){
        hiprandDestroyGenerator(gen);
        INFO_OUT("generator destroyed: ", "success");
    }

    void curand_generator(){
	std::time_t result = std::time(nullptr);
	int seed = static_cast<int>(result);
    std::cout << "time since epoch  = " << result << " (s)" << std::endl;
	CURAND_CALL(hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_MTGP32));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, seed));
	std::cout << "Curand Seed = " << seed << std::endl;

    atexit(destroy_generator);
    }

    void gen_thermal_noise(){
        CURAND_CALL(hiprandGenerateNormal(gen, gvalsx, params::Nspins, 0.0, 1.0));
        CURAND_CALL(hiprandGenerateNormal(gen, gvalsy, params::Nspins, 0.0, 1.0));
        CURAND_CALL(hiprandGenerateNormal(gen, gvalsz, params::Nspins, 0.0, 1.0));
    }
    
    __global__ void ttm(double time, int Nz, double *Te, double *Tp, double *P_it)
    {
        const int i = blockDim.x * blockIdx.x + threadIdx.x; 
        
        if (i < Nz){

            double Tep1;
            double Tpp1;
            double z;

            // if (i == 0)
            // {
                P_it[i]=c_P_0*exp(-((time-c_t0)/c_tau)*((time-c_t0)/c_tau));
                Tep1=Te[0] + (c_dt/(c_gamma_e*Te[0]))*(c_Gep*(Tp[0]-Te[0]) + P_it[0] + c_kappa_0*( (Te[0]/Tp[0]) * 2.0*(Te[1]-Te[0])*c_oneOvrdzdz));
                Tpp1=Tp[0]+(c_dt*c_Gep/c_Cp)*(Te[0]-Tp[0]);
            // }
            // if (i == Nz-1)
            // {
            //     z=static_cast<double>(Nz-1)*c_dz;
            //     P_it[Nz-1]=c_P_0*exp(-((time-c_t0)/c_tau)*((time-c_t0)/c_tau))*exp(-z/c_delta);
            //     Tep1=Te[Nz-1]+(c_dt/(c_gamma_e*Te[Nz-1]))*(c_Gep*(Tp[Nz-1]-Te[Nz-1])+P_it[Nz-1]+c_kappa_0*( (Te[Nz-1]/Tp[Nz-1]) * 2.0*(Te[Nz-2]-Te[Nz-1])*c_oneOvrdzdz));
            //     Tpp1=Tp[Nz-1]+(c_dt*c_Gep/c_Cp)*(Te[Nz-1]-Tp[Nz-1]);
            // }
            // if ((1 <= i) && (i < Nz-1))
            // {
            //     z=static_cast<double>(i)*c_dz;
            //     P_it[i]=c_P_0*exp(-((time-c_t0)/c_tau)*((time-c_t0)/c_tau))*exp(-z/c_delta);
            //     Tep1=Te[i] + (c_dt/(c_gamma_e*Te[i]))*(c_Gep*(Tp[i]-Te[i]) + P_it[i]+c_kappa_0*( (Te[i]/Tp[i]) * (Te[i+1]-2.0*Te[i]+Te[i-1])*c_oneOvrdzdz+(Tp[i]*((Te[i+1]-Te[i-1])*c_oneOvr2dz) - Te[i]*(Tp[i+1]-Tp[i-1])*c_oneOvr2dz)/(Tp[i]*Tp[i])*((Te[i+1]-Te[i-1])*c_oneOvr2dz)));
            //     Tpp1=Tp[i]+(c_dt*c_Gep/c_Cp)*(Te[i]-Tp[i]);
            // }

            //update the values of Te[i] and Tp[i]
            Te[i]=Tep1;
            Tp[i]=Tpp1;
        }
    }

    __global__ void ttf(double time, int N, double *dtfa, double *Te, int *dzlayer)
    {

        const int a = blockDim.x * blockIdx.x + threadIdx.x; 

        if (a < N){
            dtfa[a] = c_thermal_const * sqrt(Te[dzlayer[a]]);
        }
    }


    void testing(int i){

		Array<double> testingx;
		Array<double> testingy, testingz;
		testingx.resize(params::Lz);
		// testingy.resize(params::Nspins);
		// testingz.resize(params::Nspins);

		CUDA_CALL(hipMemcpy(testingx.ptr(), Te, sizeof(double) * params::Lz, hipMemcpyDeviceToHost));
		// CUDA_CALL(hipMemcpy(testingy.ptr(), Hapy, sizeof(double) * params::Lz, hipMemcpyDeviceToHost));
		// CUDA_CALL(hipMemcpy(testingz.ptr(), Hapz, sizeof(double) * params::Lz, hipMemcpyDeviceToHost));

	    std::cout << i << " ";
	    for (int a = 0; a < params::Lz; a++){
		    std::cout << testingx(a) << " ";	
	    }
	    std::cout << std::endl;
	}


    
}
